#include "hip/hip_runtime.h"
#include "utility.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "numgen.c"


#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__host__
void errorexit(const char *s) {
    printf("\n%s",s);	
    exit(EXIT_FAILURE);	 	
}

//generate elements of sequence – parallel part
__global__ 
void calculate(long inputArgument, unsigned long int* input, int *result) {
    int my_index=blockIdx.x*blockDim.x+threadIdx.x;
    
    if (my_index >= inputArgument){
      return;
    }

    unsigned long int number = input[my_index];
    if (number == 0 || number == 1){
      result[my_index] = 0;
      return;
    }
    for (long j = 2; j*j <= number; j ++ ){
      if(number % j == 0){
        result[my_index] = 0;
        return;
      }
    }
    result[my_index]=1;
}


int main(int argc,char **argv) {

  Args ins__args;
  parseArgs(&ins__args, &argc, argv);
  
  //program input argument
  long inputArgument = ins__args.arg; 
  unsigned long int *numbers = (unsigned long int*)malloc(inputArgument * sizeof(unsigned long int));
  numgen(inputArgument, numbers);

  struct timeval ins__tstart, ins__tstop;
  gettimeofday(&ins__tstart, NULL);
  

  int threadsinblock=1024;
  int blocksingrid = inputArgument / threadsinblock + 1;	

  //memory allocation on host
  int *hresults=(int*)malloc(inputArgument*sizeof(int));
  if (!hresults) errorexit("Error allocating memory on the host");	
  

  //memory allocation on device (GPU)
  unsigned long int *dinputs=NULL;
  if (hipSuccess!=hipMalloc((void **)&dinputs, inputArgument*sizeof(unsigned long int)))
    errorexit("Error allocating memory on the GPU");
  int *dresults=NULL;
  if (hipSuccess!=hipMalloc((void **)&dresults, inputArgument*sizeof(int)))
    errorexit("Error allocating memory on the GPU");

  long long result = 0;


  //copy all elements from host to device
  if (hipSuccess!=hipMemcpy(dinputs, numbers, inputArgument*sizeof(unsigned long int), hipMemcpyHostToDevice))
      errorexit("Error copying results");

  // run your CUDA kernel(s) here
  //call kernel on GPU – calculation are executed on GPU
  calculate<<<blocksingrid,threadsinblock>>>(inputArgument, dinputs, dresults);
  if (hipSuccess!=hipGetLastError())
    errorexit("Error during kernel launch");

  //copy all elements from device to host
  if (hipSuccess!=hipMemcpy(hresults, dresults, inputArgument*sizeof(int), hipMemcpyDeviceToHost))
      errorexit("Error copying results");
  
  // synchronize/finalize your CUDA computations
  for(int i=0;i<inputArgument;i++) {
    result = result + hresults[i];
  }

  printf("\nThe final result is %lld\n",result);

  //free resources
  free(hresults);
  if (hipSuccess!=hipFree(dresults))
    errorexit("Error when deallocating space on the GPU");

  gettimeofday(&ins__tstop, NULL);
  ins__printtime(&ins__tstart, &ins__tstop, ins__args.marker);
}
